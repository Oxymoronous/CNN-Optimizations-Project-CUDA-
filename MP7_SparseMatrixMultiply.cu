#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

__global__ void spmvJDSKernel(float *out, int *matColStart, int *matCols,
                              int *matRowPerm, int *matRows,
                              float *matData, float *vec, int dim) {
  //@@ insert spmv kernel for jds format
  //out = y	
  //matColStart = column index in original jds matrix
  //matCols = column indices of each element in original jds matrix
  //matRowPerm = column's original row index in original jds matrix
  //matRows = length of Columns in transposed matrix
  //matData = data stored in transposed form (all elements in the same column are elements in the same row in original jds matrix)		
  //vec = x		
  //dim = num_rows
  
  int row = blockIdx.x * blockDim.x + threadIdx.x;		//one thread calculate one output element
  if (row < dim){						//one thread calculate one element
    float dot = 0;
    //unsigned int next_index = 0;				//counter for while loop
    int i;
    for(i=0; i<matRows[row]; i++){
      dot += matData[matColStart[i]+row] * vec[matCols[matColStart[i]+row]];
    }
    out[matRowPerm[row]] = dot;
  }
}

static void spmvJDS(float *out, int *matColStart, int *matCols,
                    int *matRowPerm, int *matRows, float *matData,
                    float *vec, int dim) {

  //@@ invoke spmv kernel for jds format
	//matColStart	=	index for matCols starting point	
	//matCols	=	index of each data using column major format
	//matRowPerm	=	row index of data after sorting by most condensed to least
	//matRows	=	number of elements per row (by range)
	//matData	=	data array
	//vec		=	
  dim3 griddim(dim, 1, 1);
  dim3 blockdim(32, 1, 1);
  spmvJDSKernel<<<griddim, blockdim>>>(out, matColStart, matCols, matRowPerm, matRows, matData, vec, dim);
  return; 
}

int main(int argc, char **argv) {
  wbArg_t args;
  int *hostCSRCols;
  int *hostCSRRows;
  float *hostCSRData;
  int *hostJDSColStart;
  int *hostJDSCols;
  int *hostJDSRowPerm;
  int *hostJDSRows;
  float *hostJDSData;
  float *hostVector;
  float *hostOutput;
  int *deviceJDSColStart;
  int *deviceJDSCols;
  int *deviceJDSRowPerm;
  int *deviceJDSRows;
  float *deviceJDSData;
  float *deviceVector;
  float *deviceOutput;
  int dim, ncols, nrows, ndata;
  int maxRowNNZ;

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostCSRCols = (int *)wbImport(wbArg_getInputFile(args, 0), &ncols, "Integer");
  hostCSRRows = (int *)wbImport(wbArg_getInputFile(args, 1), &nrows, "Integer");
  hostCSRData = (float *)wbImport(wbArg_getInputFile(args, 2), &ndata, "Real");
  hostVector = (float *)wbImport(wbArg_getInputFile(args, 3), &dim, "Real");

  hostOutput = (float *)malloc(sizeof(float) * dim);

  wbTime_stop(Generic, "Importing data and creating memory on host");

  CSRToJDS(dim, hostCSRRows, hostCSRCols, hostCSRData, &hostJDSRowPerm, &hostJDSRows,
           &hostJDSColStart, &hostJDSCols, &hostJDSData);
  maxRowNNZ = hostJDSRows[0];

  wbTime_start(GPU, "Allocating GPU memory.");
  hipMalloc((void **)&deviceJDSColStart, sizeof(int) * maxRowNNZ);
  hipMalloc((void **)&deviceJDSCols, sizeof(int) * ndata);
  hipMalloc((void **)&deviceJDSRowPerm, sizeof(int) * dim);
  hipMalloc((void **)&deviceJDSRows, sizeof(int) * dim);
  hipMalloc((void **)&deviceJDSData, sizeof(float) * ndata);

  hipMalloc((void **)&deviceVector, sizeof(float) * dim);
  hipMalloc((void **)&deviceOutput, sizeof(float) * dim);
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  hipMemcpy(deviceJDSColStart, hostJDSColStart, sizeof(int) * maxRowNNZ,
             hipMemcpyHostToDevice);
  hipMemcpy(deviceJDSCols, hostJDSCols, sizeof(int) * ndata, hipMemcpyHostToDevice);
  hipMemcpy(deviceJDSRowPerm, hostJDSRowPerm, sizeof(int) * dim, hipMemcpyHostToDevice);
  hipMemcpy(deviceJDSRows, hostJDSRows, sizeof(int) * dim, hipMemcpyHostToDevice);
  hipMemcpy(deviceJDSData, hostJDSData, sizeof(float) * ndata, hipMemcpyHostToDevice);
  hipMemcpy(deviceVector, hostVector, sizeof(float) * dim, hipMemcpyHostToDevice);
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  wbTime_start(Compute, "Performing CUDA computation");
  spmvJDS(deviceOutput, deviceJDSColStart, deviceJDSCols, deviceJDSRowPerm, deviceJDSRows,
          deviceJDSData, deviceVector, dim);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  hipMemcpy(hostOutput, deviceOutput, sizeof(float) * dim, hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  hipFree(deviceVector);
  hipFree(deviceOutput);
  hipFree(deviceJDSColStart);
  hipFree(deviceJDSCols);
  hipFree(deviceJDSRowPerm);
  hipFree(deviceJDSRows);
  hipFree(deviceJDSData);

  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostOutput, dim);

  free(hostCSRCols);
  free(hostCSRRows);
  free(hostCSRData);
  free(hostVector);
  free(hostOutput);
  free(hostJDSColStart);
  free(hostJDSCols);
  free(hostJDSRowPerm);
  free(hostJDSRows);
  free(hostJDSData);

  return 0;
}
